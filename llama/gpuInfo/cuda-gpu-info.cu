#include <stddef.h>

#if defined(GPU_INFO_USE_HIPBLAS)
#include <hip/hip_runtime.h>
#include <hipblas/hipblas.h>
#define cudaGetDevice hipGetDevice
#define cudaGetDeviceCount hipGetDeviceCount
#define cudaGetErrorString hipGetErrorString
#define cudaMemGetInfo hipMemGetInfo
#define cudaSetDevice hipSetDevice
#define cudaSuccess hipSuccess
#else
#include <hip/hip_runtime.h>

#endif


typedef void (*gpuInfoErrorLogCallback_t)(const char* message);

bool gpuInfoSetCudaDevice(const int device, gpuInfoErrorLogCallback_t errorLogCallback) {
    int current_device;
    auto getDeviceResult = hipGetDevice(&current_device);

    if (getDeviceResult != hipSuccess) {
        errorLogCallback(hipGetErrorString(getDeviceResult));
        return false;
    }

    if (device == current_device) {
        return true;
    }

    const auto setDeviceResult = hipSetDevice(device);

    if (setDeviceResult != hipSuccess) {
        errorLogCallback(hipGetErrorString(setDeviceResult));
        return false;
    }

    return true;
}

bool gpuInfoGetCudaDeviceInfo(int device, size_t * total, size_t * used, gpuInfoErrorLogCallback_t errorLogCallback) {
    gpuInfoSetCudaDevice(device, errorLogCallback);

    size_t freeMem;
    size_t totalMem;
    auto getMemInfoResult = hipMemGetInfo(&freeMem, &totalMem);

    if (getMemInfoResult != hipSuccess) {
        errorLogCallback(hipGetErrorString(getMemInfoResult));
        return false;
    }

    *total = totalMem;
    *used = totalMem - freeMem;

    return true;
}

int gpuInfoGetCudaDeviceCount(gpuInfoErrorLogCallback_t errorLogCallback) {
    int deviceCount;
    auto getDeviceCountResult = hipGetDeviceCount(&deviceCount);

    if (getDeviceCountResult != hipSuccess) {
        errorLogCallback(hipGetErrorString(getDeviceCountResult));
        return -1;
    }

    return deviceCount;
}

bool gpuInfoGetTotalCudaDevicesInfo(size_t * total, size_t * used, gpuInfoErrorLogCallback_t errorLogCallback) {
    int deviceCount = gpuInfoGetCudaDeviceCount(errorLogCallback);

    if (deviceCount < 0) {
        return false;
    }

    size_t usedMem = 0;
    size_t totalMem = 0;

    for (int i = 0; i < deviceCount; i++) {
        size_t deviceUsedMem;
        size_t deviceTotalMem;

        if (!gpuInfoGetCudaDeviceInfo(i, &deviceTotalMem, &deviceUsedMem, errorLogCallback)) {
            return false;
        }

        usedMem += deviceUsedMem;
        totalMem += deviceTotalMem;
    }

    *total = totalMem;
    *used = usedMem;

    return true;
}
